#include "hip/hip_runtime.h"
// @file vl_nnpool.cu
// @brief Pooling block MEX wrapper
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-15 Andrea Vedaldi and Karel Lenc.
Copyright (C) 2017 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "bits/mexutils.h"
#include "bits/datamex.hpp"
#include "bits/nnpooling.hpp"

#if ENABLE_GPU
#include "bits/datacu.hpp"
#endif

#include <cassert>

using Int = vl::Int ;

/* option codes */
enum {
  opt_stride = 0,
  opt_padding,
  opt_shape,
  opt_method,
  opt_verbose,
  opt_cudnn,
  opt_no_cudnn,
} ;

/* options */
VLMXOption  options [] = {
  {"Stride",           1,   opt_stride            },
  {"Pad",              1,   opt_padding           },
  {"Padding",          1,   opt_padding           },
  {"Shape",            1,   opt_shape             },
  {"Method",           1,   opt_method            },
  {"Verbose",          0,   opt_verbose           },
  {"CUDNN",            0,   opt_cudnn             },
  {"NoCUDNN",          0,   opt_no_cudnn          },
  {0,                  0,   0                     }
} ;

/* ---------------------------------------------------------------- */
/*                                                          Context */
/* ---------------------------------------------------------------- */

vl::MexContext context ;

void atExit()
{
  context.clear() ;
}

/* ---------------------------------------------------------------- */
/*                                                       MEX driver */
/* ---------------------------------------------------------------- */

enum {
  IN_DATA = 0, IN_SIZE, IN_DEROUTPUT, IN_END
} ;

enum {
  OUT_RESULT = 0, OUT_END
} ;

vl::ErrorCode
performPooling(vl::MexContext& context,
               int nout, mxArray *out[],
               int nin, mxArray const *in[])
{
  bool backMode = false ;

  int verbosity = 0 ;
  int opt ;
  int next = IN_END ;
  mxArray const *optarg ;

  /* -------------------------------------------------------------- */
  /*                                            Check the arguments */
  /* -------------------------------------------------------------- */

  if (nin < 2) {
    return context.setError(vl::VLE_IllegalArgument, "There are less than two arguments.") ;
  }

  if (nin > 2 && vlmxIsString(in[2],-1)) {
    next = 2 ;
    backMode = 0 ;
  } else {
    backMode = (nin >= 3) ;
  }

  vl::nn::Pooling op(context) ;

  {
    optarg = in[IN_SIZE] ;
    MXOPTIVEC(SHAPE,setShape)
  }

  while ((opt = vlmxNextOption (in, nin, options, &next, &optarg)) >= 0) {
    switch (opt) {
      case opt_verbose : context.setLogLevel(++verbosity) ; break ;
      case opt_stride : MXOPTIVEC(STRIDE,setStride) ; break ;
      case opt_padding : MXOPTIVEC(PADDING,setPadding) ; break ;
      case opt_shape: MXOPTIVEC(SHAPE,setShape) ; break ;
      case opt_method :
        if (!vlmxIsString(optarg,-1)) {
           return context.setError(vl::VLE_IllegalArgument, "METHOD is not a string.") ;
        }
        if (vlmxIsEqualToStringI(optarg, "max")) {
          op.setMethod(vl::nn::Pooling::Max) ;
        } else if (vlmxIsEqualToStringI(optarg, "avg")) {
          op.setMethod(vl::nn::Pooling::Average) ;
        } else {
          return context.setError(vl::VLE_IllegalArgument,
                                  "The value of METHOD is not a supported method.") ;
        }
        break;

      case opt_no_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(false) ;
#endif
        break ;

      case opt_cudnn :
#if ENABLE_CUDNN
        context.getCudaHelper().setCudnnEnabled(true) ;
#endif
        break ;

      default:
        break ;
    }
  }

  vl::MexTensor data(context) ;
  data.init(in[IN_DATA]) ;
  data.reshape(4) ;

  if (!backMode) {
    // Forward mode.
    vl::DeviceType deviceType = data.getDeviceType() ;
    vl::DataType dataType = data.getDataType() ;

    // Compute the size of the output tensor.
    vl::TensorShape outputShape ;
    MXCHECK(op.forwardShape(outputShape,data)) ;

    // Initialize output tensor.
    vl::MexTensor output(context) ;
    output.initWithZeros(deviceType, dataType, outputShape) ;

    // Perform calculation.
    MXCHECK(op.forward(output,data)) ;

    // Return results.
    out[OUT_RESULT] = output.relinquish() ;
  }
  else {
    // Backward mode.
    vl::MexTensor derOutput(context) ;
    derOutput.init(in[IN_DEROUTPUT]) ;
    derOutput.reshape(4) ;
    vl::DeviceType deviceType = derOutput.getDeviceType() ;
    vl::DataType dataType = derOutput.getDataType() ;

    // Initialize the tensors to be returned.
    vl::MexTensor derData(context) ;
    derData.initWithZeros(deviceType, dataType, data.getShape()) ;
    derData.reshape(4) ;

    // Perform calculation.
    MXCHECK(op.backward(derData,data,derOutput)) ;

    // Return results.
    out[OUT_RESULT] = derData.relinquish() ;
  }

  return vl::VLE_Success ;
}

void mexFunction(int nout, mxArray *out[],
                 int nin, mxArray const *in[])
{
  mexAtExit(atExit) ;
  context.setLogLevel(0) ;
  context.clearLog() ;

  vl::ErrorCode error = performPooling(context,nout,out,nin,in) ;

  if (context.getLogLevel() > 0) {
    mexPrintf("vl_nnpool:\n") ;
    for (auto const & str : context.getLogbook()) {
      mexPrintf("\t%s\n", str.c_str()) ;
    }
    context.setLogLevel(0) ;
  }

  if (error != vl::VLE_Success) {
    vlmxError(VLMXE_IllegalArgument, context.getLastErrorMessage().c_str()) ;
  }
  return ;
}

