#include "hip/hip_runtime.h"
// @file nnroipooling_gpu.cu
// @brief ROI pooling block (GPU)
// @author Hakan Bilen
// @author Abishek Dutta
// @author Andrea Vedaldi

/*
Copyright (C) 2016-17 Hakan Bilen, Abishek Dutta, and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "datacu.hpp"

#include <cassert>
#include <cfloat>
#include <algorithm>
#include <sm_20_atomic_functions.h>

using Int = vl::Int ;

// -------------------------------------------------------------------
//                                                             Helpers
// -------------------------------------------------------------------

template<typename T>
struct Geom {
  int subdivisions[2] ; // GPU uses int not Int
  T transform[6] ;
  Geom(std::array<Int,2> const &subdivisions, std::array<double,6> const &transform)
  {
    this->subdivisions[0] = (int)subdivisions[0] ;
    this->subdivisions[1] = (int)subdivisions[1] ;
    for (size_t i = 0 ; i < 6 ; ++i) this->transform[i] = (T)transform[i] ;
  }
} ;

struct Bounds {
  int image, offset, hstart, hend, wstart, wend ;
  bool isEmpty ;
} ;

template<typename T>
__device__ __forceinline__ static Bounds
getBounds(int outputIndex,
          int height, int width, int numChannels, int cardinality,
          const T* rois, int numROIs,
          Geom<T> geom)
{
  Bounds b ;

  int ph = outputIndex ;
  int pw = ph / geom.subdivisions[0] ;
  int pc = pw / geom.subdivisions[1] ;
  int pr = pc / numChannels ;

  ph %= geom.subdivisions[0] ;
  pw %= geom.subdivisions[1] ;
  pc %= numChannels ;

  rois += 5 * pr ;

  // Apply sacle and offset to each ROI coordinate.
  T u1_ = rois[1] ;
  T v1_ = rois[2] ;
  T u2_ = rois[3] ;
  T v2_ = rois[4] ;

  T u1 = geom.transform[0] * u1_ + geom.transform[2] * v1_ + geom.transform[4] ;
  T v1 = geom.transform[1] * u1_ + geom.transform[3] * v1_ + geom.transform[5] ;
  T u2 = geom.transform[0] * u2_ + geom.transform[2] * v2_ + geom.transform[4] ;
  T v2 = geom.transform[1] * u2_ + geom.transform[3] * v2_ + geom.transform[5] ;

  // First and last pixel of each ROI (rounded
  // for compatibility with the Caffe definition).
  int roi_image   = (int)rois[0];
  int roi_start_h = (int)::round(v1) - 1 ;
  int roi_start_w = (int)::round(u1) - 1 ;
  int roi_end_h   = (int)::round(v2) - 1 ;
  int roi_end_w   = (int)::round(u2) - 1 ;
  int roi_height  = max(roi_end_h - roi_start_h + 1, 1) ;
  int roi_width   = max(roi_end_w - roi_start_w + 1, 1) ;

  T bin_size_h = (T)roi_height / geom.subdivisions[0] ;
  T bin_size_w = (T)roi_width / geom.subdivisions[1] ;

  roi_image = min(max(roi_image - 1,0), (int)cardinality - 1) ;
  b.offset = (roi_image * numChannels + pc) * (width*height) ;

  b.wstart = (int)floor(((T)pw) * bin_size_w) ;
  b.wend = (int)ceil(((T)(pw + 1)) * bin_size_w) ;
  b.wstart = min(max(b.wstart + roi_start_w, 0), (int)width) ;
  b.wend = min(max(b.wend + roi_start_w, 0), (int)width) ;

  b.hstart = (int)floor(((T)ph) * bin_size_h) ;
  b.hend = (int)ceil(((T)(ph + 1)) * bin_size_h) ;
  b.hstart = min(max(b.hstart + roi_start_h, 0), (int)height) ;
  b.hend = min(max(b.hend + roi_start_h, 0), (int)height) ;

  b.isEmpty = (b.hend <= b.hstart) || (b.wend <= b.wstart) ;

  return b ;
}

/* ---------------------------------------------------------------- */
/*                                       roipooling_average_forward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
roipooling_average_kernel
(T* output,
 const T* data, int height, int width, int numChannels, int cardinality,
 const T* rois, int numROIs,
 Geom<T> geom)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  int outputVolume = geom.subdivisions[0] * geom.subdivisions[1] * numChannels * numROIs;
  if (outputIndex < outputVolume) {
    Bounds b = getBounds<T>(outputIndex,
                            height,width,numChannels,cardinality,
                            rois,numROIs,
                            geom) ;
    data += b.offset ;
    T bestValue = 0;
    const T coeff = ((T)1.) / (T)((b.wend-b.wstart) * (b.hend-b.hstart));
    for (int w = b.wstart; w < b.wend; ++w) {
      for (int h = b.hstart; h < b.hend; ++h) {
        int index = w * height + h ;
        bestValue += data[index] * coeff ;
      }
    }
    output[outputIndex] = bestValue ;
  }
}

/* ---------------------------------------------------------------- */
/*                                           roipooling_max_forward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
roipooling_max_kernel
(T* output,
 const T* data, int height, int width, int numChannels, int cardinality,
 const T* rois, int numROIs,
 Geom<T> geom)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x ;
  int outputVolume = geom.subdivisions[0] * geom.subdivisions[1] * numChannels * numROIs ;
  if (outputIndex < outputVolume) {
    Bounds b = getBounds<T>(outputIndex,
                            height,width,numChannels,cardinality,
                            rois,numROIs,
                            geom) ;
    data += b.offset ;
    if (! b.isEmpty) {
      T bestValue = -FLT_MAX;
      for (int w = b.wstart; w < b.wend; ++w) {
        for (int h = b.hstart; h < b.hend; ++h) {
          int index = w * height + h ;
          bestValue = max(bestValue, data[index]) ;
        }
      }
      output[outputIndex] = bestValue ;
    } else {
      output[outputIndex] = 0 ;
    }
  }
}

/* ---------------------------------------------------------------- */
/*                                      roipooling_average_backward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
roipooling_average_backward_kernel
(T* derData,
 const T* data, int height, int width, int numChannels, int cardinality,
 const T* rois, int numROIs,
 const T* derOutput,
 Geom<T> geom)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  int outputVolume = geom.subdivisions[0] * geom.subdivisions[1] * numChannels * numROIs ;
  if (outputIndex < outputVolume) {

    Bounds b = getBounds<T>(outputIndex,
                            height,width,numChannels,cardinality,
                            rois,numROIs,
                            geom) ;
    data += b.offset ;
    derData += b.offset ;
    const T coeff = ((T)1.) / (T)((b.wend-b.wstart)*(b.hend-b.hstart)) ;
    for (int h = b.hstart; h < b.hend; ++h) {
      for (int w = b.wstart; w < b.wend; ++w) {
        int index = w * height + h ;
        atomicAdd(derData + index, derOutput[outputIndex] * coeff) ;
      }
    }
  }
}

/* ---------------------------------------------------------------- */
/*                                          roipooling_max_backward */
/* ---------------------------------------------------------------- */

template<typename T> __global__ void
roipooling_max_backward_kernel
(T* derData,
 const T* data, int height, int width, int numChannels, int cardinality,
 const T* rois, int numROIs,
 const T* derOutput,
 Geom<T> geom)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  int outputVolume = geom.subdivisions[0] * geom.subdivisions[1] * numChannels * numROIs;

  if (outputIndex < outputVolume) {

    Bounds b = getBounds<T>(outputIndex,
                            height,width,numChannels,cardinality,
                            rois,numROIs,
                            geom) ;
    if (! b.isEmpty) {
      data += b.offset ;
      derData += b.offset ;
      int bestIndex = min(b.wstart,width-1) * height + min(b.hstart,height-1);
      T bestValue = -FLT_MAX;
      for (int h = b.hstart; h < b.hend; ++h) {
        for (int w = b.wstart; w < b.wend; ++w) {
          int index = w * height + h ;
          T value = data[index] ;
          if (value > bestValue) {
            bestValue = value ;
            bestIndex = index ;
          }
        }
      }
      atomicAdd(derData + bestIndex, derOutput[outputIndex]) ;
    }
  }
}

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType, ROIPooling::Method method>
struct ROIPoolingForwardGPU
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &output,
                           Tensor const &input,
                           Tensor const &rois)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int numROIs = rois.getNumElements() / 5 ;
    Int outputVolume = op.getSubdivisions()[0] * op.getSubdivisions()[1] * input.getNumChannels() * numROIs ;

    auto func = roipooling_max_kernel<type> ;
    if (method == ROIPooling::Average) {
      func = roipooling_average_kernel<type> ;
    }
    else { assert(method == ROIPooling::Max) ; }

    func <<< divideAndRoundUp((unsigned)outputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
    ((type*)output.getMemory(),
     (type const*)input.getMemory(),
     (int)input.getHeight(),(int)input.getWidth(),(int)input.getNumChannels(),(int)input.getCardinality(),
     (type const*)rois.getMemory(), (int)numROIs,
     Geom<type>(op.getSubdivisions(),op.getTransform())) ;

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;

template<DataType dataType>
struct ROIPoolingForward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor pooled,
                           Tensor input,
                           Tensor rois)
  {
    switch (op.getMethod()) {
      case ROIPooling::Max:
        return ROIPoolingForwardGPU<dataType,ROIPooling::Max>
        ()(op,pooled,input,rois) ;
      case ROIPooling::Average:
        return ROIPoolingForwardGPU<dataType,ROIPooling::Average>
        ()(op,pooled,input,rois) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType, ROIPooling::Method method>
struct ROIPoolingBackwardGPU
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int numROIs = rois.getNumElements() / 5 ;
    Int outputVolume = op.getSubdivisions()[0] * op.getSubdivisions()[1] * input.getNumChannels() * numROIs ;

    auto func = roipooling_max_backward_kernel<type> ;
    if (method == ROIPooling::Average) {
      func = roipooling_average_backward_kernel<type> ;
    }
    else { assert(method == ROIPooling::Max) ; }

    func <<< divideAndRoundUp((unsigned)outputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
    ((type*)derInput.getMemory(),
     (type const*)input.getMemory(),
     (int)input.getHeight(), (int)input.getWidth(), (int)input.getNumChannels(), (int)input.getCardinality(),
     (type const*)rois.getMemory(), (int)numROIs,
     (type const*)derOutput.getMemory(),
     Geom<type>(op.getSubdivisions(),op.getTransform())) ;

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;

template<DataType dataType>
struct ROIPoolingBackward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    switch (op.getMethod()) {
      case ROIPooling::Max:
        return ROIPoolingBackwardGPU<dataType,ROIPooling::Max>
        ()(op,derInput,input,rois,derOutput) ;
      case ROIPooling::Average:
        return ROIPoolingBackwardGPU<dataType,ROIPooling::Average>
        ()(op,derInput,input,rois,derOutput) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

