// @file nnbias_cudnn.cu
// @brief biasolution block CuDNN-based implementation.
// @author Andrea Vedaldi

/*
Copyright (C) 2015-17 Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnbias.hpp"
#include "datacu.hpp"
#include "impl/cudnnhelper.hpp"
#include <cassert>

using namespace std ;
using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = op.getContext().setError(op.getContext().getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__LINE__) ":" STRINGIZE(__FILE__))) ; \
goto done ; \
} }

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType>
struct BiasForwardCudnn
{
  vl::ErrorCode operator()(Bias const& op,
                           Tensor &output, double outputMult,
                           Tensor const &input, double inputMult,
                           Tensor const &bias, double biasMult)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t outputDesc, biasDesc, dataDesc ;
    bool outputDescInitialized = false ;
    bool biasDescInitialized = false ;
    bool dataDescInitialized = false ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Get output tensor descripotr
    assert(output) ;
    CHECK(hipdnnCreateTensorDescriptor(&outputDesc)) ;
    outputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(outputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     DataTypeToCudnn<dataType>::dataType,
                                     (int)output.getSize(), // sizes
                                     (int)output.getDepth(),
                                     (int)output.getWidth(),
                                     (int)output.getHeight())) ;

    if (bias) {
      CHECK(hipdnnCreateTensorDescriptor(&biasDesc)) ;
      biasDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(biasDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       1,
                                       (int)bias.getNumElements(),
                                       1,
                                       1)) ;

      auto alpha = static_cast<type>(biasMult) ;
      auto beta = static_cast<type>(outputMult) ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_SAME_C,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           outputDesc, output.getMemory())) ;
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           outputDesc, output.getMemory())) ;
#endif
      outputMult = 1 ;
    }

    if (input) {
      CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
      dataDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       (int)input.getSize(),
                                       (int)input.getDepth(),
                                       (int)input.getWidth(),
                                       (int)input.getHeight())) ;

      auto alpha = static_cast<type>(biasMult) ;
      auto beta = static_cast<type>(outputMult) ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_FULL_TENSOR,
                           &alpha,
                           dataDesc, input.getMemory(),
                           &beta,
                           outputDesc, output.getMemory()));
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           dataDesc, input.getMemory(),
                           &beta,
                           outputDesc, output.getMemory()));
#endif
    }

    /* cleanup */
  done:
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (biasDescInitialized) { hipdnnDestroyTensorDescriptor(biasDesc) ; }
    if (outputDescInitialized) { hipdnnDestroyTensorDescriptor(outputDesc) ; }
    return op.getContext().passError(error, __func__) ;
  }
} ;


// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType>
struct BiasBackwardCudnn
{
  vl::ErrorCode operator()(Bias const &op,
                           Tensor &derInput, double derInputMult,
                           Tensor &derBias, double derBiasMult,
                           double inputMult, double biasMult,
                           Tensor const &derOutput)
  {
    typedef typename DataTypeTraits<dataType>::type type ;

    /* no derInputDesc needed as same as dataDesc */
    hipdnnTensorDescriptor_t derInputDesc, derBiasDesc, derOutputDesc ;
    bool derInputDescInitialized = false ;
    bool derBiasDescInitialized = false ;
    bool derOutputDescInitialized = false ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Must have derOutput for all derivatives
    assert(derOutput) ;
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     DataTypeToCudnn<dataType>::dataType,
                                     (int)derOutput.getSize(), // sizes
                                     (int)derOutput.getDepth(),
                                     (int)derOutput.getWidth(),
                                     (int)derOutput.getHeight())) ;

    // for derivatives w.r.t. bias
    if (derBias) {
      CHECK(hipdnnCreateTensorDescriptor(&derBiasDesc)) ;
      derBiasDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(derBiasDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       1,
                                       (int)derBias.getNumElements(),
                                       1,
                                       1)) ;

      auto alpha = static_cast<type>(biasMult) ;
      auto beta = static_cast<type>(derBiasMult) ;
      CHECK(hipdnnConvolutionBackwardBias
            (handle,
             &alpha,
             derOutputDesc, (type const*)derOutput.getMemory(),
             &beta,
             derBiasDesc, (type*)derBias.getMemory())) ;
    }

    if (derInput) {
      CHECK(hipdnnCreateTensorDescriptor(&derInputDesc)) ;
      derInputDescInitialized = true ;
      CHECK(hipdnnSetTensor4dDescriptor(derInputDesc,
                                       HIPDNN_TENSOR_NCHW,
                                       DataTypeToCudnn<dataType>::dataType,
                                       (int)derInput.getSize(),
                                       (int)derInput.getDepth(),
                                       (int)derInput.getWidth(),
                                       (int)derInput.getHeight())) ;
      auto alpha = static_cast<type>(biasMult) ;
      auto beta = static_cast<type>(derBiasMult) ;
#if (CUDNN_VERSION < 4000)
      CHECK(hipdnnAddTensor(handle,
                           CUDNN_ADD_SAME_C,
                           &alpha,
                           biasDesc, bias.getMemory(),
                           &beta,
                           derInputDesc, derInput.getMemory())) ;
#else
      CHECK(hipdnnAddTensor(handle,
                           &alpha,
                           derOutputDesc, derOutput.getMemory(),
                           &beta,
                           derInputDesc, derInput.getMemory())) ;
#endif
    }

  done:
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    if (derBiasDescInitialized) { hipdnnDestroyTensorDescriptor(derBiasDesc) ; }
    if (derInputDescInitialized) { hipdnnDestroyTensorDescriptor(derInputDesc) ; }
    return op.getContext().passError(error, __func__) ;
  }
} ;
