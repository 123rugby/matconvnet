#include "hip/hip_runtime.h"
// @file nnpooling_gpu.cu
// @brief Pooling block GPU.
// @author Andrea Vedaldi
// @author Karel Lenc

/*
Copyright (C) 2014-17 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnpooling.hpp"
#include "datacu.hpp"
#include <cassert>
#include <float.h>
#include <sm_20_atomic_functions.h>

// -------------------------------------------------------------------
//                                                 Max pooling helpers
// -------------------------------------------------------------------

template<typename T> __global__ void
pooling_max_kernel
(T* output,
 const T* data,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    data += pz * (width*height) ;

    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;

    T bestValue = data[y1 * width + x1] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        bestValue = max(bestValue, data[y * width + x]) ;
      }
    }
    output[outputIndex] = bestValue ;
  }
}

#ifdef VLNN_CAFFELIKE_BPPOOL
// In order to be able to use this, BP would need to have access to both
// bottom data and output data (currently only passed bottom data...)
template <typename T> __global__ void
pooling_max_backward_with_output_data
(T* derData,
 const T* data,
 const T* output,
 const T* derOutput,
 const int nthreads,
 const int outputWidth,
 const int outputHeight,
 const int width,
 const int height,
 const int depth,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int x = index % width;
    int y = (index / width) % height;
    int z = (index / width / height) % depth;
    int py1 = (y < poolHeight) ? 0 : (y - poolHeight) / strideY + 1;
    int py2 = min(y / strideY + 1, outputHeight);
    int px1 = (x < poolWidth) ? 0 : (x - poolWidth) / strideX + 1;
    int px2 = min(x / strideX + 1, outputWidth);
    T gradient = 0;
    T datum = data[(z * height + y) * width + x];
    output += z * outputHeight * outputWidth;
    dzdy += z * outputHeight * outputWidth;
    for (int py = py1; py < py2; ++py) {
      for (int px = px1; px < px2; ++px) {
        gradient += dzdy[py * outputWidth + px] *
        (datum == output[py * outputWidth + px]);
      }
    }
    dzdx[index] = gradient;
  }
}
#endif

template<typename T> __global__ void
pooling_max_backward_kernel
(T* derData,
 const T* data,
 const T* derOutput,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    data += pz * (width*height) ;
    derData += pz * (width*height) ;

    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;

    int bestIndex = y1 * width + x1 ;
    T bestValue = data[bestIndex] ;
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        int index = y * width + x ;
        T value = data[index] ;
        if (value > bestValue) {
          bestValue = value ;
          bestIndex = index ;
        }
      }
    }
    /*
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    atomicAdd(derData + bestIndex, derOutput[outputIndex]) ;
  }
}

// -------------------------------------------------------------------
//                                             Average pooling helpers
// -------------------------------------------------------------------

template<typename T> __global__ void
pooling_average_kernel
(T* output,
 const T* data,
 const int outputWidth,
 const int outputHeight,
 const int outputVolume,
 const int width,
 const int height,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  /* outputIndex = x + y * outputWidth + z * (outputWidth * outputHeight) */
  int outputIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (outputIndex < outputVolume) {
    int px = outputIndex ;
    int py = px / outputWidth ;
    int pz = py / outputHeight ;
    px %= outputWidth ;
    py %= outputHeight ;
    int x1 = px * strideX - padLeft ;
    int y1 = py * strideY - padTop ;
    int x2 = min(x1 + poolWidth, width) ;
    int y2 = min(y1 + poolHeight, height) ;
    x1 = max(x1, 0) ;
    y1 = max(y1, 0) ;
    data += pz * (width*height) ;
    T accum = 0;
    T poolSize = (y2 - y1)*(x2 - x1);
    for (int y = y1 ; y < y2 ; ++y) {
      for (int x = x1 ; x < x2 ; ++x) {
        accum += data[y * width + x] ;
      }
    }
    output[outputIndex] = accum / poolSize ;
  }
}

template <typename T> __global__ void
pooling_average_backward_kernel
(T* derData,
 const T* derOutput,
 const int nthreads,
 const int outputWidth,
 const int outputHeight,
 const int width,
 const int height,
 const int depth,
 const int poolWidth,
 const int poolHeight,
 const int strideX,
 const int strideY,
 const int padLeft,
 const int padTop)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < nthreads) {
    /* To understand the logic of this piece of code see the
     comments to of the row2im backward kernel */
    int x_data = index ;
    int y_data = x_data / width ;
    int z = y_data / height ;
    x_data %= width ;
    y_data %= height ;

    int dx = x_data + padLeft - poolWidth ;
    int dy = y_data + padTop - poolHeight ;
    int px1 = (dx >= 0) ? dx/strideX + 1 : 0 ;
    int py1 = (dy >= 0) ? dy/strideY + 1 : 0 ;
    int px2 = min((x_data + padLeft) / strideX, outputWidth - 1) ;
    int py2 = min((y_data + padTop) / strideY, outputHeight - 1) ;
    T accumulator = 0 ;
    derOutput += z * outputHeight * outputWidth;
    for (int py = py1 ; py <= py2 ; ++py) {
      for (int px = px1 ; px <= px2 ; ++px) {
        int x1 = px * strideX - padLeft ;
        int y1 = py * strideY - padTop ;
        int x2 = min(x1 + poolWidth, width) ;
        int y2 = min(y1 + poolHeight, height) ;
        x1 = max(x1, 0) ;
        y1 = max(y1, 0) ;
        T poolSize = (y2 - y1) * (x2 - x1);
        accumulator += derOutput[py * outputWidth + px] / poolSize ;
      }
    }
    derData[index] = accumulator ;
  }
}

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType, Pooling::Method method>
struct PoolingForwardGPU
{
  vl::ErrorCode operator()(Pooling const &op,
                           Tensor &output,
                           Tensor const &input)
  {
    // Argument sanity check.
    assert(output) ;
    assert(input) ;
    TensorShape outShape ;
    op.forwardShape(outShape, input) ;
    assert(outShape == output) ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int outputVolume = output.getNumElements() ;

    if (method == Pooling::Max) {
      pooling_max_kernel<type>
      <<< divideAndRoundUp((unsigned)outputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      ((type*)output.getMemory(),
       (type const*)input.getMemory(),
       (int)output.getHeight(), (int)output.getWidth(), (int)output.getNumElements(),
       (int)input.getHeight(), (int)input.getWidth(),
       (int)op.getShape(0), (int)op.getShape(1),
       (int)op.getStride(0), (int)op.getStride(1),
       (int)op.getPadding(0), (int)op.getPadding(2));
    }
    else if (method == Pooling::Average) {
      pooling_average_kernel<type>
      <<< divideAndRoundUp((unsigned)outputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      ((type*)output.getMemory(),
       (type const*)input.getMemory(),
       (int)output.getHeight(), (int)output.getWidth(), (int)output.getNumElements(),
       (int)input.getHeight(), (int)input.getWidth(),
       (int)op.getShape(0), (int)op.getShape(1),
       (int)op.getStride(0), (int)op.getStride(1),
       (int)op.getPadding(0), (int)op.getPadding(2));
    }
    else {
      assert(false) ;
    }

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ;

template<DataType dataType>
struct PoolingForward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(Pooling const &op,
                           Tensor output,
                           Tensor input)
  {
    switch (op.getMethod()) {
      case Pooling::Max:
        return
        PoolingForwardGPU<dataType,Pooling::Max>
        ()(op,output,input) ;
      case Pooling::Average:
        return
        PoolingForwardGPU<dataType,Pooling::Average>
        ()(op,output,input) ;
      default:
        return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType, Pooling::Method method>
struct PoolingBackwardGPU
{
  vl::ErrorCode operator()(Pooling const &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &derOutput)
  {
    // Argument sanity check.
    assert(derInput) ;
    assert(derOutput) ;
    TensorShape outputShape ;
    op.forwardShape(outputShape,derInput) ;
    assert(derOutput == outputShape) ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int outputVolume = derOutput.getNumElements() ;
    Int inputVolume = input.getNumElements() ;

    if (method == Pooling::Max) {
      assert(input) ;
      pooling_max_backward_kernel<type>
      <<< divideAndRoundUp((unsigned)outputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      ((type*)derInput.getMemory(),
       (type const*)input.getMemory(),
       (type const*)derOutput.getMemory(),
       (int)derOutput.getHeight(), (int)derOutput.getWidth(), (int)outputVolume,
       (int)input.getHeight(), (int)input.getWidth(),
       (int)op.getShape(0), (int)op.getShape(1),
       (int)op.getStride(0), (int)op.getStride(1),
       (int)op.getPadding(0), (int)op.getPadding(2));
    }
    else if (method == Pooling::Average) {
      pooling_average_backward_kernel<type>
      <<< divideAndRoundUp((unsigned)inputVolume,VL_CUDA_NUM_THREADS),VL_CUDA_NUM_THREADS >>>
      ((type*)derInput.getMemory(),
       (type const*)derOutput.getMemory(),
       (int)inputVolume,
       (int)derOutput.getHeight(), (int)derOutput.getWidth(),
       (int)input.getHeight(), (int)input.getWidth(), (int)(input.getDepth()*input.getSize()),
       (int)op.getShape(0), (int)op.getShape(1),
       (int)op.getStride(0), (int)op.getStride(1),
       (int)op.getPadding(0), (int)op.getPadding(2)) ;
    }
    else {
      assert(false) ;
    }

    hipError_t status = hipPeekAtLastError() ;
    return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
  }
} ; // pooling_max


template<DataType dataType>
struct PoolingBackward<VLDT_GPU,dataType>
{
  vl::ErrorCode operator()(Pooling const &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &derOutput)
  {
    switch (op.getMethod()) {
      case Pooling::Max:
        return
        PoolingBackwardGPU<dataType,Pooling::Max>
        ()(op,derInput,input,derOutput) ;
      case Pooling::Average:
        return
        PoolingBackwardGPU<dataType,Pooling::Average>
        ()(op,derInput,input,derOutput) ;
      default:
        return VLE_IllegalArgument ;
    }
  }
} ;
