#include "hip/hip_runtime.h"
// @file nnroipooling.cu
// @brief ROI pooling block
// @author Hakan Bilen
// @author Abishek Dutta
// @author Andrea Vedaldi

/*
Copyright (C) 2016-17 Hakan Bilen, Abishek Dutta, and Andrea Vedaldi.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnroipooling.hpp"
#include "impl/dispatcher.hpp"
#include <limits>
#include <cassert>
#include <cstring>
#include <cmath>
#include <iostream>

using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

template<DeviceType deviceType, DataType dataType> struct ROIPoolingForward ;
template<DeviceType deviceType, DataType dataType> struct ROIPoolingBackward ;

#if ENABLE_GPU
#include "nnroipooling_gpu.cu"
#endif

// -------------------------------------------------------------------
//                                                             Helpers
// -------------------------------------------------------------------

template <typename type>
struct acc_max
{
  inline acc_max(Int poolHeight, Int poolWidth, type derOutput = 0)
  :
  value(-std::numeric_limits<type>::infinity()),
  derOutput(derOutput),
  derDataActivePt(NULL)
  { }

  inline void accumulate_forward(type x) {
    value = std::max(value, x) ;
  }

  inline void accumulate_backward(type const* data, type* derDataPt) {
    type x = *data ;
    if (x > value) {
      value = x ;
      derDataActivePt = derDataPt ;
    }
  }

  inline type done_forward() const {
    return value ;
  }

  inline void done_backward() const {
    if (derDataActivePt) { *derDataActivePt += derOutput ; }
  }

  type value ;
  type derOutput ;
  type* derDataActivePt ;
} ;

template <typename type>
struct acc_sum
{
  inline acc_sum(Int poolHeight, Int poolWidth, type derOutput = 0)
  :
  value(0),
  scale(type(1)/type(poolHeight*poolWidth)),
  derOutput(derOutput)
  { }

  inline void accumulate_forward(type x) {
    value += x ;
  }

  inline void accumulate_backward(type const* data, type* derDataPt) {
    *derDataPt += derOutput * scale ;
  }

  inline type done_forward() const {
    return value * scale ;
  }

  inline void done_backward() const { }

  type value ;
  type derOutput ;
  type scale;
} ;

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType, class Accumulator>
struct ROIPoolingForwardCPU
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &pooled,
                           Tensor const &input,
                           Tensor const &rois)
  {
    static const std::string signature = std::string("ROIPoolingForward[MCN,")
    + DeviceTypeTraits<VLDT_CPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int numROIs = rois.getNumElements() / 5 ;
    Int height = input.getHeight() ;
    Int width = input.getWidth() ;
    Int depth = input.getNumChannels() ;
    Int size = input.getCardinality() ;
    auto roisData = (type const*)rois.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto pooledData = (type*)pooled.getMemory() ;

    // For each ROI R = [t x1 y1 x2 y2].
    for (Int roi = 0; roi < numROIs; ++roi) {

      // Apply scale and offset to each ROI coordinate.
      type u1_ = roisData[5 * roi + 1] ;
      type v1_ = roisData[5 * roi + 2] ;
      type u2_ = roisData[5 * roi + 3] ;
      type v2_ = roisData[5 * roi + 4] ;

      auto const& tf = op.getTransform() ;
      type u1 = (type)tf[0] * u1_ + (type)tf[2] * v1_ + (type)tf[4] ;
      type v1 = (type)tf[1] * u1_ + (type)tf[3] * v1_ + (type)tf[5] ;
      type u2 = (type)tf[0] * u2_ + (type)tf[2] * v2_ + (type)tf[4] ;
      type v2 = (type)tf[1] * u2_ + (type)tf[3] * v2_ + (type)tf[5] ;

      // First and last pixel of each ROI (rounded
      // for compatibility with the Caffe definition).
      Int roi_image   = (Int)roisData[5 * roi + 0];
      Int roi_start_h = (Int)::round(v1) - 1 ;
      Int roi_start_w = (Int)::round(u1) - 1 ;
      Int roi_end_h   = (Int)::round(v2) - 1 ;
      Int roi_end_w   = (Int)::round(u2) - 1 ;
      Int roi_height  = std::max(roi_end_h - roi_start_h + 1, (Int)1) ;
      Int roi_width   = std::max(roi_end_w - roi_start_w + 1, (Int)1) ;

      roi_image = std::min(std::max(roi_image - 1, (Int)0), size - 1) ;
      type const * data_offset = inputData + (roi_image * depth) * (width*height) ;

      type bin_size_h = (type)roi_height / op.getSubdivisions()[0] ;
      type bin_size_w = (type)roi_width / op.getSubdivisions()[1] ;

      // For each feature channel.
      for (Int z = 0; z < depth; ++z) {

        // For each column of tiles.
        for (Int pw = 0; pw < op.getSubdivisions()[1]; ++pw) {
          Int wstart = (Int)floor(((type)pw) * bin_size_w) ;
          Int wend = (Int)ceil(((type)(pw + 1)) * bin_size_w) ;
          wstart = std::min(std::max(wstart + roi_start_w, (Int)0), width) ;
          wend = std::min(std::max(wend + roi_start_w, (Int)0), width) ;

          // For each tile in a column.
          for (Int ph = 0; ph < op.getSubdivisions()[0]; ++ph) {
            Int hstart = (Int)floor(((type)ph) * bin_size_h) ;
            Int hend = (Int)ceil(((type)(ph + 1)) * bin_size_h) ;
            hstart = std::min(std::max(hstart + roi_start_h, (Int)0), height) ;
            hend = std::min(std::max(hend + roi_start_h, (Int)0), height) ;

            bool is_empty = (hend <= hstart) || (wend <= wstart);

            if (is_empty) {
              *pooledData++ = 0 ;
            }
            else {
              Accumulator acc(hend - hstart, wend - wstart) ;
              for (Int w = wstart ; w < wend; ++w) {
                for (Int h = hstart ; h < hend; ++h) {
                  auto const index = w * height + h ;
                  acc.accumulate_forward(data_offset[index]) ;
                }
              }
              *pooledData++ = acc.done_forward() ;
            }
          } // end of ph
        } // end of pw
        data_offset += width*height;
      } // end of z
    } // end of n
    return VLE_Success ;
  }
} ;

template<DataType dataType>
struct ROIPoolingForward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &pooled,
                           Tensor const &input,
                           Tensor const &rois)
  {
    switch (op.getMethod()) {
      case ROIPooling::Max:
        return
        ROIPoolingForwardCPU<dataType,acc_max<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,pooled,input,rois) ;
      case ROIPooling::Average:
        return
        ROIPoolingForwardCPU<dataType,acc_sum<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,pooled,input,rois) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType, class Accumulator>
struct ROIPoolingBackwardCPU
{
  vl::ErrorCode operator()(ROIPooling const &op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    static const std::string signature = std::string("ROIPoolingBackward[MCN,")
    + DeviceTypeTraits<VLDT_CPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int numROIs = rois.getNumElements() / 5 ;
    Int height = input.getHeight() ;
    Int width = input.getWidth() ;
    Int depth = input.getNumChannels() ;
    Int size = input.getCardinality() ;
    
    auto derInputData = (type*)derInput.getMemory() ;
    auto roisData = (type const*)rois.getMemory() ;
    auto inputData = (type const*)input.getMemory() ;
    auto derOutputData = (type const*)derOutput.getMemory() ;

    memset(derInputData, 0, (size_t)derInput.getNumElements() * sizeof(type)) ;

    // For each ROI R = [t x1 y1 x2 y2].
    for (Int roi = 0; roi < numROIs ; ++roi) {

      // Apply sacle and offset to each ROI coordinate.
      type u1_ = roisData[5 * roi + 1] ;
      type v1_ = roisData[5 * roi + 2] ;
      type u2_ = roisData[5 * roi + 3] ;
      type v2_ = roisData[5 * roi + 4] ;

      auto const& tf = op.getTransform() ;
      type u1 = (type)tf[0] * u1_ + (type)tf[2] * v1_ + (type)tf[4] ;
      type v1 = (type)tf[1] * u1_ + (type)tf[3] * v1_ + (type)tf[5] ;
      type u2 = (type)tf[0] * u2_ + (type)tf[2] * v2_ + (type)tf[4] ;
      type v2 = (type)tf[1] * u2_ + (type)tf[3] * v2_ + (type)tf[5] ;

      // First and last pixel of each ROI (rounded
      // for compatibility with the Caffe definition).
      Int roi_image   = (Int)roisData[5 * roi + 0];
      Int roi_start_h = (Int)::round(v1) - 1 ;
      Int roi_start_w = (Int)::round(u1) - 1 ;
      Int roi_end_h   = (Int)::round(v2) - 1 ;
      Int roi_end_w   = (Int)::round(u2) - 1 ;
      Int roi_height = std::max(roi_end_h - roi_start_h + 1, (Int)1) ;
      Int roi_width = std::max(roi_end_w - roi_start_w + 1, (Int)1) ;

      roi_image = std::min(std::max(roi_image - 1,(Int)0), size - 1) ;
      type const * data_offset = inputData + roi_image * (depth*width*height) ;
      type * derInputData_offset = derInputData + roi_image * (depth*width*height) ;

      const type bin_size_h = (type)roi_height / op.getSubdivisions()[0] ;
      const type bin_size_w = (type)roi_width / op.getSubdivisions()[1] ;

      // For each feature channel.
      for (Int z = 0; z < depth; ++z) {

        // For each column of tiles.
        for (Int pw = 0; pw < op.getSubdivisions()[1]; ++pw) {
          Int wstart = (Int)floor(((type)pw) * bin_size_w) ;
          Int wend = (Int)ceil(((type)(pw + 1)) * bin_size_w) ;
          wstart = std::min(std::max(wstart + roi_start_w, (Int)0), width) ;
          wend = std::min(std::max(wend + roi_start_w, (Int)0), width) ;

          // For each tile in a column.
          for (Int ph = 0; ph < op.getSubdivisions()[0]; ++ph) {
            Int hstart = (Int)floor(((type)ph) * bin_size_h) ;
            Int hend = (Int)ceil(((type)(ph + 1)) * bin_size_h) ;
            hstart = std::min(std::max(hstart + roi_start_h, (Int)0), height) ;
            hend = std::min(std::max(hend + roi_start_h, (Int)0), height) ;

            Accumulator acc(hend - hstart, wend - wstart, *derOutputData++) ;
            for (Int w = wstart; w < wend; ++w) {
              for (Int h = hstart; h < hend; ++h) {
                auto const index = w * height + h ;
                acc.accumulate_backward(&data_offset[index],
                                        &derInputData_offset[index]) ;
              }
            }
            acc.done_backward() ;
          } // end of pw
        } // end of ph
        data_offset += width*height ;
        derInputData_offset += width*height ;
      } // end of z
    } // end of n

    return VLE_Success ;
  }
} ;

template<DataType dataType>
struct ROIPoolingBackward<VLDT_CPU,dataType>
{
  vl::ErrorCode operator()(ROIPooling const&op,
                           Tensor &derInput,
                           Tensor const &input,
                           Tensor const &rois,
                           Tensor const &derOutput)
  {
    switch (op.getMethod()) {
      case ROIPooling::Max: return
        ROIPoolingBackwardCPU<dataType,acc_max<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,derInput,input,rois,derOutput) ;
      case ROIPooling::Average: return
        ROIPoolingBackwardCPU<dataType,acc_sum<typename vl::DataTypeTraits<dataType>::type> >
        ()(op,derInput,input,rois,derOutput) ;
      default: return VLE_IllegalArgument ;
    }
  }
} ;

// -------------------------------------------------------------------
/// MARK: - Driver
// -------------------------------------------------------------------

ROIPooling::ROIPooling(Context &context,
                       std::vector<Int> const& subdivisions,
                       std::vector<double> const& transform,
                       Method method) :
Operation(context),
subdivisions(subdivisions),
transform(transform),
method(method)
{ }

ROIPooling::ROIPooling(Context &context)
:
Operation(context),
subdivisions {1,1},
transform {1., 0., 0., 1., 0., 0.},
method (Max)
{ }

ErrorCode ROIPooling::setSubdivisions(std::vector<Int> const& subdivisions) {
  // Stride must be positive.
  if (any_of(begin(subdivisions),end(subdivisions),[](Int x){return x <= 0;})) {
    return getContext().setError
    (VLE_IllegalArgument, "An element of SUBDIVISIONS is less than 1.") ;
  }
  // There must one stride per spatial dimension.
  if (Int(subdivisions.size()) == getNumSpatialDimensions()) {
    this->subdivisions = subdivisions ;
  }
  else if (subdivisions.size() == 1) {
    fill(begin(this->subdivisions),end(this->subdivisions),subdivisions[0]) ;
  }
  else {
    return getContext().setError
    (VLE_IllegalArgument, "SUBDIVISIONS is neither scalar nor has the same"
     " cardinality as the number of spatial dimensions.") ;
  }
  return VLE_Success ;
}

ErrorCode ROIPooling::setTransform(std::vector<double> const& transform)
{
  // There must one stride per spatial dimension.
  Int ns = getNumSpatialDimensions() ;
  if (Int(transform.size()) == (ns)*(ns+1)) {
    this->transform = transform ;
  }
  else if ((Int)transform.size() == 2*ns) {
    fill(begin(this->transform),end(this->transform),.0) ;
    for (Int i = 0 ; i < ns ; ++i) {
      this->transform[size_t(i + ns*i)] = transform[size_t(i)] ;
      this->transform[size_t(i + ns*ns)] = transform[size_t(i+ns)] ;
    }
  }
  else if (transform.size() == 1) {
    fill(begin(this->transform),end(this->transform),.0) ;
    for (Int i = 0 ; i < ns ; ++i) {
      this->transform[size_t(i + ns*i)] = transform[0] ;
    }
  }
  else {
    return getContext().setError
    (VLE_IllegalArgument, "TRANSFORMS is neither scalar nor has the the "
     "appropriate size for the number of spatial dimensions.") ;
  }
  return VLE_Success ;
}

ErrorCode ROIPooling::setMethod(Method method) {
  if (method != Average && method != Max) {
    return getContext().setError(VLE_IllegalArgument, "Unknown METHOD.") ;
  }
  this->method = method ;
  return VLE_Success ;
}

vl::ErrorCode
ROIPooling::forwardShape(TensorShape &output,
                         TensorShape const& input,
                         TensorShape const& rois) const
{
  output.clear() ;
  auto ns = getNumSpatialDimensions() ;

  // INPUT must have spatial dimensions, channels, and instances.
  if (input.getNumDimensions() > ns+2) {
    return getContext().setError
    (VLE_TensorShapeMismatch, "ROIPooling: INPUT has too many dimensions.") ;
  }

  // ROIS must contain an integer number of ROI specifications.
  Int numROIs = rois.getNumElements() / 5 ;
  if (numROIs * 5 != rois.getNumElements()) {
    return getContext().setError
    (VLE_TensorShapeMismatch, "ROIPooling: the number of elements of ROI is not a multiple of 5.") ;
  }

  // Output has size SUBD... x INPUT_CHANNELS x NUMROIS.
  std::vector<Int> dims (size_t(ns + 2)) ;
  copy(begin(subdivisions),end(subdivisions),begin(dims)) ;
  dims[size_t(ns)] = input.getDimension(ns) ;
  dims[size_t(ns)+1] = numROIs ;

  output = dims ;
  return VLE_Success ;
}

vl::ErrorCode
ROIPooling::forward(Tensor &output,
                    Tensor const &input,
                    Tensor const &rois) const
{
  // Validate arguments.
  ErrorCode error ;
  if (!check_tensor_compatibility(output,input,rois)) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: the tensors have mismatching data or device type.") ;
  }
  TensorShape outputShape ;
  if ((error = forwardShape(outputShape, input, rois)) != VLE_Success) {
    return error ;
  }
  if (output != outputShape) {
    return getContext().setError
    (VLE_TensorShapeMismatch,
     "ROIPoolingForward: OUTPUT does not have the appropriate dimensions.") ;
  }
  if (input.isEmpty() || input.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: INPUT is empty or null.") ;
  }
  if (input.isEmpty() || input.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: OUTPUT is empty or null.") ;
  }
  if (rois.isEmpty() || rois.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: ROI is empty or null.") ;
  }

  VLLOG(*this,1)
  << "ROIPoolingForward:"
  << " subdivisions=" << pretty(getSubdivisions())
  << " transform=" << pretty(getTransform()) ;

  VLLOG(*this,1)
  << "ROIPoolingForward:"
  << " input=" << pretty(input.getDimensions())
  << " rois=" << pretty(rois.getDimensions())
  << " output=" << pretty(output.getDimensions()) ;

  return getContext().passError
  (dispatch<ROIPoolingForward>()(*this,output,input,rois),
   "ROIPoolingForward") ;
}

vl::ErrorCode
ROIPooling::backward(Tensor &derInput,
                     Tensor const &input,
                     Tensor const &rois,
                     Tensor const &derOutput) const
{
  // Validate arguments.
  ErrorCode error ;
  if (!check_tensor_compatibility(derInput,input,rois,derOutput)) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: the tensors have mismatching data or device type.") ;
  }
  TensorShape outputShape ;
  if ((error = forwardShape(outputShape, input, rois)) != VLE_Success) {
    return error ;
  }
  if (derOutput != outputShape) {
    return getContext().setError
    (VLE_TensorShapeMismatch,
     "ROIPoolingForward: OUTPUT does not have the appropriate dimensions.") ;
  }
  if (input.isEmpty() || input.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: INPUT is empty or null.") ;
  }
  if (input.isEmpty() || input.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: OUTPUT is empty or null.") ;
  }
  if (rois.isEmpty() || rois.isNull()) {
    return getContext().setError
    (VLE_IllegalArgument,
     "ROIPoolingForward: ROI is empty or null.") ;
  }

  VLLOG(*this,1)
  << "ROIPoolingBackward:"
  << " subdivisions=" << pretty(getSubdivisions())
  << " transform=" << pretty(getTransform())
  << " method=" << (getMethod() == Average ? "Average" : "Max") ;

  VLLOG(*this,1)
  << "ROIPoolingBackward:"
  << " derInput=" << pretty(derInput.getDimensions())
  << " input=" << pretty(input.getDimensions())
  << " rois=" << pretty(rois.getDimensions())
  << " derOutput=" << pretty(derOutput.getDimensions()) ;

  return getContext().passError
  (dispatch<ROIPoolingBackward>()(*this,derInput,input,rois,derOutput),
   "ROIPoolingBackward") ;
}





