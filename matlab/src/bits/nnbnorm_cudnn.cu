#include "hip/hip_runtime.h"
// @file nnbnorm_cudnn.hpp
// @brief bnorm CuDNN-based implementation.
// @author Andrea Vedaldi

/*
 Copyright (C) 2016-17 Andrea Vedaldi.
 All rights reserved.

 This file is part of the VLFeat library and is made available under
 the terms of the BSD license (see the COPYING file).
 */

#include "nnbnorm.hpp"
#include "datacu.hpp"
#include "impl/cudnnhelper.hpp"
#include "impl/copy.hpp"
#include <cassert>

using namespace std ;
using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

#define CHECK(x) \
{ \
cudnnError = x ; \
if (cudnnError != HIPDNN_STATUS_SUCCESS) { \
error = op.getContext().setError(op.getContext().getCudaHelper().catchCudnnError(cudnnError, \
STRINGIZE(__FILE__) ":" STRINGIZE(__LINE__))) ; \
goto done ; \
} }

// -------------------------------------------------------------------
//                                                             Kernels
// -------------------------------------------------------------------

template<typename T>
__global__ void var_to_std(T * var, unsigned int num, T scale, T epsilon)
{
  unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx < num) {
    var[idx] = sqrt(scale * var[idx] + epsilon) ;
  }
}

template<typename T>
__global__ void std_to_var(T * var, T const * std, unsigned int num, T epsilon)
{
  unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx < num) {
    var[idx] = std[idx]*std[idx] - epsilon ;
  }
}

template<typename T>
__global__ void inverse(T * ivar, unsigned int num)
{
  unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx < num) {
    ivar[idx] = ((T)1) / ivar[idx] ;
  }
}

template<typename T>
__global__ void inverse(T * out, T * in, unsigned int num)
{
  unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;
  if (idx < num) {
    out[idx] = ((T)1) / in[idx] ;
  }
}

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<DataType dataType>
struct BatchNormForwardWithMomentCudnn
{
  vl::ErrorCode operator()(BatchNorm const &op,
                           Tensor &output,
                           Tensor const &moment, // can be null
                           Tensor const &input,
                           Tensor const &multiplier,
                           Tensor const &bias)
  {
    static const std::string signature = std::string("BatchNormForwardWithMoment[CuDNN,")
    + DeviceTypeTraits<VLDT_GPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    if (op.getEpsilon() < HIPDNN_BN_MIN_EPSILON) { return VLE_Unsupported ; }

    assert(output) ;
    assert(input) ;
    assert(multiplier) ;
    assert(bias) ;

    typedef typename DataTypeTraits<dataType>::type type ;
    size_t workspaceSize ;
    type * workspace ;

    hipdnnTensorDescriptor_t dataDesc, momentDesc ;
    bool dataDescInitialized = false ;
    bool momentDescInitialized = false ;

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN.
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs.
    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     (int)input.getCardinality(),
                                     (int)input.getNumChannels(),
                                     (int)input.getWidth(),
                                     (int)input.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&momentDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(momentDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     1, (int)input.getNumChannels(), 1, 1)) ;

    // Allocate workspace.
    workspaceSize = (size_t)input.getNumChannels() ;
    workspace = (type*)op.getContext().getWorkspace(vl::VLDT_GPU, workspaceSize * sizeof(type)) ;

    // Run CuDNN batch normalization implementation.
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      type * meanMemory = moment ? (type*)moment.getMemory() : workspace ;
      type * stdMemory = meanMemory + input.getNumChannels() ;
      type * varMemory = workspace ;

      auto blockSize = VL_CUDA_NUM_THREADS ;
      std_to_var<type>
      <<< divideAndRoundUp((unsigned)input.getNumChannels(),blockSize),blockSize >>>
      (varMemory, stdMemory, (unsigned)input.getNumChannels(), (type)HIPDNN_BN_MIN_EPSILON) ;

      CHECK(hipdnnBatchNormalizationForwardInference
            (handle,
             HIPDNN_BATCHNORM_SPATIAL,
             &alpha,
             &beta,
             dataDesc, input.getMemory(),
             dataDesc, output.getMemory(),
             momentDesc, multiplier.getMemory(), bias.getMemory(),
             meanMemory, varMemory, HIPDNN_BN_MIN_EPSILON)) ;
    }

    // Finish.
  done:
    if (momentDescInitialized) { hipdnnDestroyTensorDescriptor(momentDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    return op.getContext().passError(error, signature.c_str()) ;
  }
} ;

template<DataType dataType>
struct BatchNormForwardCudnn
{
  vl::ErrorCode operator()(BatchNorm const &op,
                           Tensor &output,
                           Tensor &moment,
                           Tensor const &input,
                           Tensor const &multiplier,
                           Tensor const &bias)
  {
    static const std::string signature = std::string("BatchNormForward[CuDNN,")
    + DeviceTypeTraits<VLDT_GPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    if (op.getEpsilon() < HIPDNN_BN_MIN_EPSILON) { return VLE_Unsupported ; }

    assert(output) ;
    assert(input) ;
    assert(multiplier) ;
    assert(bias) ;

    typedef typename DataTypeTraits<dataType>::type type ;

    hipdnnTensorDescriptor_t dataDesc, momentDesc ;
    bool dataDescInitialized = false ;
    bool momentDescInitialized = false ;

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = output.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN.
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs.
    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     (int)input.getCardinality(),
                                     (int)input.getNumChannels(),
                                     (int)input.getWidth(),
                                     (int)input.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&momentDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(momentDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     1, (int)input.getNumChannels(), 1, 1)) ;

    // Run CuDNN batch normalization implementation.
    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      type * meanMemory = NULL ;
      type * varMemory = NULL ;

      if (moment) {
        meanMemory = (type*)moment.getMemory()  ;
        varMemory = meanMemory + input.getNumChannels() ;
        vl::impl::operations<vl::VLDT_GPU,type>::fill
        (meanMemory, 2 * size_t(input.getNumChannels()) * sizeof(type), 0) ;
      }

      CHECK(hipdnnBatchNormalizationForwardTraining
            (handle,
             HIPDNN_BATCHNORM_SPATIAL,
             &alpha, &beta,
             dataDesc, input.getMemory(),
             dataDesc, output.getMemory(),
             momentDesc, multiplier.getMemory(), bias.getMemory(),
             0, NULL, NULL,
             op.getEpsilon(),
             meanMemory, varMemory)) ;

      if (varMemory) {
        // CuDNN computes the variance without epsilon, whereas MCN
        // returns the standard deviation after adding epsilon.
        // Also, CuDNN returns the unbiased variance estimate, but it is
        // debatable that this is appropriate.
        //
        // We pick instead the caches, which are closer to the values we compute.
        // Also they do not need to be pre-initialized with zeros.

        auto blockSize = VL_CUDA_NUM_THREADS ;
        inverse<type>
        <<< divideAndRoundUp((unsigned)input.getNumChannels(),blockSize),blockSize >>>
        (varMemory, (unsigned)input.getNumChannels()) ;
      }
    }

    // Finish.
  done:
    if (momentDescInitialized) { hipdnnDestroyTensorDescriptor(momentDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    return op.getContext().passError(error,signature.c_str()) ;
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<DataType dataType>
struct BatchNormBackwardWithMomentCudnn
{
  vl::ErrorCode operator()(BatchNorm const &op,
                           Tensor &derInput,
                           Tensor &derMultiplier,
                           Tensor &derBias,
                           Tensor const &moment,
                           Tensor const &input,
                           Tensor const &multiplier,
                           Tensor const &bias,
                           Tensor const &derOutput)
  {
    static const std::string signature = std::string("BatchNormBackwardWithMoment[CuDNN,")
    + DeviceTypeTraits<VLDT_GPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    if (op.getEpsilon() < HIPDNN_BN_MIN_EPSILON) { return VLE_Unsupported ; }

    assert(derInput) ;
    assert(derMultiplier) ;
    assert(derBias) ;
    assert(moment) ;
    assert(input) ;
    assert(multiplier) ;
    assert(bias) ;
    assert(derOutput) ;

    typedef typename DataTypeTraits<dataType>::type type ;
    size_t workspaceSize ;
    type * workspace ;

    hipdnnTensorDescriptor_t derOutputDesc, dataDesc, momentDesc ;
    bool derOutputDescInitialized = false ;
    bool dataDescInitialized = false ;
    bool momentDescInitialized = false ;

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = derOutput.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN.
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs.
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     (int)derOutput.getCardinality(), // sizes
                                     (int)derOutput.getNumChannels(),
                                     (int)derOutput.getWidth(),
                                     (int)derOutput.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&dataDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(dataDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     (int)input.getCardinality(),
                                     (int)input.getNumChannels(),
                                     (int)input.getWidth(),
                                     (int)input.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&momentDesc)) ;
    dataDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(momentDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     1, (int)input.getNumChannels(), 1, 1)) ;


    // Scrarch space to provide moments in CuDNN format.
    workspaceSize = (size_t)derInput.getNumChannels() ;
    workspace = (type*)op.getContext().getWorkspace(vl::VLDT_GPU, workspaceSize * sizeof(type)) ;

    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      type * meanMemory = (type*)moment.getMemory() ;
      type * stdMemory = meanMemory + input.getNumChannels() ;
      type * istdMemory = workspace ;

      // The CuDNN manual describes the varMemory output above
      // as inverse variance, but it is the inverse standard deviation instead.
      auto blockSize = VL_CUDA_NUM_THREADS ;
      inverse<type>
      <<< divideAndRoundUp((unsigned)input.getNumChannels(),blockSize),blockSize >>>
      (istdMemory, stdMemory, (unsigned)input.getNumChannels()) ;

      CHECK(hipdnnBatchNormalizationBackward
            (handle,
             HIPDNN_BATCHNORM_SPATIAL,
             &alpha, &beta, // data
             &alpha, &beta, // params
             dataDesc, input.getMemory(), // input
             derOutputDesc, derOutput.getMemory(), // input
             dataDesc, derInput.getMemory(), // output
             momentDesc, multiplier.getMemory(), // input
             derMultiplier.getMemory(), // output
             derBias.getMemory(), // output
             op.getEpsilon(),
             meanMemory, istdMemory)) ;
    }

    // Finish.
  done:
    if (momentDescInitialized) { hipdnnDestroyTensorDescriptor(momentDesc) ; }
    if (dataDescInitialized) { hipdnnDestroyTensorDescriptor(dataDesc) ; }
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    return op.getContext().passError(error, signature.c_str()) ;
  }
} ;

template<DataType dataType>
struct BatchNormBackwardCudnn
{
  vl::ErrorCode operator()(BatchNorm const &op,
                           Tensor &derInput,
                           Tensor &derMultiplier,
                           Tensor &derBias,
                           Tensor &moment,
                           Tensor const &input,
                           Tensor const &multiplier,
                           Tensor const &bias,
                           Tensor const &derOutput)
  {
    static const std::string signature = std::string("BatchNormBackward[CuDNN,")
    + DeviceTypeTraits<VLDT_GPU>::name + "," + DataTypeTraits<dataType>::name + "]" ;
    VLLOG(op,1) << signature ;

    if (op.getEpsilon() < HIPDNN_BN_MIN_EPSILON) { return VLE_Unsupported ; }

    assert(derInput) ;
    assert(derMultiplier) ;
    assert(derBias) ;
    assert(input) ;
    assert(multiplier) ;
    assert(bias) ;
    assert(derOutput) ;

    typedef typename DataTypeTraits<dataType>::type type ;
    size_t workspaceSize ;
    type * workspace ;
    size_t volume ;

    hipdnnTensorDescriptor_t derOutputDesc, momentDesc ;
    bool derOutputDescInitialized = false ;
    bool momentDescInitialized = false ;

    hipdnnDataType_t cudnnDataType = DataTypeToCudnn<dataType>::dataType ;
    vl::DataType dynDataType = derOutput.getDataType() ;
    assert(dynDataType == dataType) ;

    hipdnnStatus_t cudnnError = HIPDNN_STATUS_SUCCESS ;
    vl::ErrorCode error = vl::VLE_Success ;
    hipdnnHandle_t handle ;

    // Get CuDNN.
    CHECK(op.getContext().getCudaHelper().getCudnnHandle(&handle)) ;

    // Get tensor descripotrs.
    CHECK(hipdnnCreateTensorDescriptor(&derOutputDesc)) ;
    derOutputDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(derOutputDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     (int)derOutput.getCardinality(), // sizes
                                     (int)derOutput.getNumChannels(),
                                     (int)derOutput.getWidth(),
                                     (int)derOutput.getHeight())) ;

    CHECK(hipdnnCreateTensorDescriptor(&momentDesc)) ;
    momentDescInitialized = true ;
    CHECK(hipdnnSetTensor4dDescriptor(momentDesc,
                                     HIPDNN_TENSOR_NCHW,
                                     cudnnDataType,
                                     1, (int)input.getNumChannels(), 1, 1)) ;

    // Compute moment using CuDNN. Unfortunately CuDNN does not expose
    // the values of the moment in the backward pass, so we need to run
    // the forward code to get them.

    volume = (size_t)derInput.getNumElements() ;
    workspaceSize = (moment ? 0 : size_t(2 * derInput.getNumChannels()) + volume) ;
    workspace = (type*)op.getContext().getWorkspace(vl::VLDT_GPU, workspaceSize * sizeof(type)) ;

    {
      type alpha = 1.0f ;
      type beta = 0.0f ;
      type * outMemory = workspace ;
      type * meanMemory = moment ? (type*)moment.getMemory() : workspace + volume ;
      type * varMemory = meanMemory + input.getNumChannels() ;

      CHECK(hipdnnBatchNormalizationForwardTraining
            (handle,
             HIPDNN_BATCHNORM_SPATIAL,
             &alpha, &beta,
             derOutputDesc, input.getMemory(),
             derOutputDesc, outMemory, // will be discarded
             momentDesc, multiplier.getMemory(), bias.getMemory(),
             1.0, // cumulative factor for moment
             NULL, NULL,
             op.getEpsilon(),
             meanMemory, varMemory)) ;

      CHECK(hipdnnBatchNormalizationBackward
            (handle,
             HIPDNN_BATCHNORM_SPATIAL,
             &alpha, &beta, // data
             &alpha, &beta, // params
             derOutputDesc, input.getMemory(), // input
             derOutputDesc, derOutput.getMemory(), // input
             derOutputDesc, derInput.getMemory(), // output
             momentDesc, multiplier.getMemory(), // input
             derMultiplier.getMemory(), // output
             derBias.getMemory(), // output
             op.getEpsilon(),
             meanMemory, varMemory)) ;

      // The CuDNN manual describes the varMemory output above
      // as inverse variance, but it is the inverse standard deviation instead.
      auto blockSize = VL_CUDA_NUM_THREADS ;
      inverse<type>
      <<< divideAndRoundUp((unsigned)input.getNumChannels(),blockSize),blockSize >>>
      (varMemory, (unsigned)input.getNumChannels()) ;
    }

    // Finish.
  done:
    if (momentDescInitialized) { hipdnnDestroyTensorDescriptor(momentDesc) ; }
    if (derOutputDescInitialized) { hipdnnDestroyTensorDescriptor(derOutputDesc) ; }
    return op.getContext().passError(error, signature.c_str()) ;
  }
} ;

