#include "hip/hip_runtime.h"
// @file nnsubsample.cu
// @brief Subsampling block
// @author Andrea Vedaldi

/*
Copyright (C) 2014-17 Andrea Vedaldi and Karel Lenc.
All rights reserved.

This file is part of the VLFeat library and is made available under
the terms of the BSD license (see the COPYING file).
*/

#include "nnsubsample.hpp"
#include "impl/dispatcher.hpp"
#include "impl/blashelper.hpp"
#include <cassert>
#include <cstring>

using namespace std ;
using namespace vl ;
using namespace vl::nn ;
using namespace vl::impl ;

template<vl::DeviceType deviceType, vl::DataType dataType> struct SubsampleForward ;
template<vl::DeviceType deviceType, vl::DataType dataType> struct SubsampleBackward ;

// -------------------------------------------------------------------
//                                                             Forward
// -------------------------------------------------------------------

template<vl::DataType dataType>
struct SubsampleForward<vl::VLDT_CPU, dataType>
{
  vl::ErrorCode operator()(Subsample const &op,
                           Tensor &output,
                           Tensor const &input)
  {
    // Argument sanity check.
    assert(output) ;
    assert(input) ;
    TensorShape outShape ;
    op.forwardShape(outShape, input) ;
    assert(outShape == output) ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    Int width = input.getWidth() ;
    Int height = input.getHeight() ;
    Int depth = input.getNumChannels() ;
    Int size = input.getCardinality() ;
    auto inputData = (type*)input.getMemory() ;
    auto outputData = (type*)output.getMemory() ;

    Int outputHeight = output.getHeight() ;
    Int outputWidth = output.getWidth() ;
    Int strideY = op.getStride(0) ;
    Int strideX = op.getStride(1) ;
    Int padTop = op.getPadding(0) ;
    Int padLeft = op.getPadding(2) ;

    for (Int z = 0; z < depth * size ; ++z) {
      for (Int x = 0; x < outputWidth ; ++x) {
        for (Int y = 0; y < outputHeight ; ++y) {
          auto x1 = x * strideX - padLeft ;
          auto y1 = y * strideY - padTop ;
          type value = 0 ;
          if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
            value = inputData[x1 * height + y1] ;
          }
          outputData[x * outputHeight + y] = value ;
        }
      }
      inputData += width*height ;
      outputData += outputWidth*outputHeight ;
    }
    return VLE_Success ;
  }
} ;

template<vl::DeviceType deviceType, vl::DataType dataType>
struct SubsampleAndBiasForward
{
  vl::ErrorCode operator()(Subsample const &op,
                           Tensor &output,
                           Tensor const &input,
                           Tensor const &biases)
  {
    assert(output) ;
    assert(input) ;

    vl::ErrorCode error ;
    typedef typename vl::DataTypeTraits<dataType>::type type ;

    error = SubsampleForward<deviceType,dataType>()(op,output,input) ;
    if (error != VLE_Success) { return error ; }

    auto numOutputPixels = output.getHeight() * output.getWidth() ;
    type const* allOnesMemory = (type*) op.getContext().getAllOnes(deviceType, dataType, (size_t)numOutputPixels) ;

    if (allOnesMemory == NULL) {
      error = op.getContext().getLastError() ;
      goto done ;
    }

    for (Int image = 0 ; image < input.getCardinality() ; ++image) {
      auto outputOffset = (output.getHeight()*output.getWidth()*output.getNumChannels()) * image ;
      if (biases) {
        type alpha = 1 ;
        type beta = 1 ;
        error = vl::impl::blas<deviceType, dataType>::gemm
        (op.getContext(),
         'n', 'n',
         numOutputPixels,
         biases.getNumElements(), 1,
         alpha,
         allOnesMemory, numOutputPixels,
         (type*)biases.getMemory(), 1,
         beta,
         (type*)output.getMemory() + outputOffset, numOutputPixels) ;
        if (error != vl::VLE_Success) { goto done ; }
      }
    }
  done:
    return op.getContext().passError(error, __func__) ;
  }
} ;

// -------------------------------------------------------------------
//                                                            Backward
// -------------------------------------------------------------------

template<vl::DataType dataType>
struct SubsampleBackward<vl::VLDT_CPU, dataType>
{
  vl::ErrorCode operator()(Subsample const &op,
                           Tensor &derInput,
                           Tensor const &derOutput)
  {
    assert(derInput) ;
    assert(derOutput) ;

    typedef typename vl::DataTypeTraits<dataType>::type type ;
    auto width = derInput.getWidth() ;
    auto height = derInput.getHeight() ;
    auto depth = derInput.getNumChannels() ;
    auto size = derInput.getCardinality() ;
    auto derInputData = (type*)derInput.getMemory() ;
    auto derOutputData = (type*)derOutput.getMemory() ;

    // Check argument compatibility
    TensorShape outShape ;
    op.forwardShape(outShape, derInput) ;
    assert(outShape == derOutput) ;
    Int outputHeight = derOutput.getHeight() ;
    Int outputWidth = derOutput.getWidth() ;
    Int strideY = op.getStride(0) ;
    Int strideX = op.getStride(1) ;
    Int padTop = op.getPadding(0) ;
    Int padLeft = op.getPadding(2) ;

    memset(derInputData, 0, sizeof(type) * size_t(width * height * depth * size)) ;

    for (Int z = 0; z < depth * size; ++z) {
      for (Int px = 0; px < outputWidth; ++px) {
        for (Int py  = 0; py < outputHeight; ++py) {
          auto x1 = px * strideX - padLeft ;
          auto y1 = py * strideY - padTop ;
          if (x1 >= 0 && x1 < width && y1 >= 0 && y1 < height) {
            derInputData[x1 * height + y1]
            = derOutputData[px * outputHeight + py] ;
          }
        }
      }
      derInputData += width*height ;
      derOutputData += outputWidth*outputHeight ;
    }
    return VLE_Success ;
  }
} ;

template<vl::DeviceType deviceType, vl::DataType dataType>
struct SubsampleAndBiasBackward
{
  vl::ErrorCode operator()(vl::nn::Subsample const &op,
                           vl::Tensor derInput,
                           vl::Tensor derBiases,
                           vl::Tensor derOutput)
  {
    assert(derOutput) ;

    vl::ErrorCode error = VLE_Success ;
    typedef typename vl::DataTypeTraits<dataType>::type type ;

    // Compute derInput.
    if (derInput) {
      error = SubsampleBackward<deviceType,dataType>()(op,derInput,derOutput) ;
      if (error != VLE_Success) { return error ; }
    }

    // Compute derBiases.
    if (derBiases) {
      auto numOutputPixels = derOutput.getHeight() * derOutput.getWidth() ;
      type const* allOnesMemory = (type*) op.getContext().getAllOnes(deviceType, dataType, (size_t)numOutputPixels) ;

      if (allOnesMemory == NULL) {
        error = op.getContext().getLastError() ;
        goto done ;
      }

      for (Int image = 0 ; image < derInput.getCardinality() ; ++image) {
        auto derOutputOffset = (derOutput.getHeight()*derOutput.getWidth()*derOutput.getNumChannels()) * image ;
        type alpha = 1 ;
        type beta = (image > 0) ; // Avoids having to clear derOutputs first.
        error = vl::impl::blas<deviceType,dataType>::gemv
        (op.getContext(),
         't',
         numOutputPixels, derOutput.getNumChannels(),
         alpha,
         (type const*)derOutput.getMemory() + derOutputOffset, numOutputPixels,
         allOnesMemory, 1,
         beta,
         (type*)derBiases.getMemory(), 1) ;
        if (error != vl::VLE_Success) { goto done ; }
      }
    }

  done:
    return op.getContext().passError(error, __func__) ;
  }
} ;

// -------------------------------------------------------------------
//                                                              Driver
// -------------------------------------------------------------------

#if ENABLE_GPU
#include "nnsubsample_gpu.cu"
#endif

Subsample::Subsample(vl::Context &context,
                     Int strideY, Int strideX,
                     Int padTop, Int padBottom,
                     Int padLeft, Int padRight)
: ConvolutionLike(context,2)
{
  setStride({strideY, strideX}) ;
  setPadding({padTop, padBottom, padLeft, padRight}) ;
}

vl::ErrorCode
Subsample::forwardWithBias(vl::Tensor &output,
                           vl::Tensor const &input,
                           vl::Tensor const &biases) const
{
  return dispatch<SubsampleAndBiasForward>()(*this,output,input,biases) ;
}

vl::ErrorCode
Subsample::forwardShape(vl::TensorShape &output, vl::TensorShape const& input) const
{
  output = TensorShape() ; // null
  if (input.getNumDimensions() < getNumSpatialDimensions()) {
    return VLE_IllegalArgument ;
  }
  output = input ;
  for (Int d = 0 ; d < getNumSpatialDimensions() ; ++d) {
    auto odim = convLikeSizeHelper(input.getDimension(d),
                                   1,
                                   getStride(d),
                                   {getPadding(2*d),getPadding(2*d+1)},
                                   1) ;
    output.setDimension(d, odim) ;
  }
  return VLE_Success ;
}

vl::ErrorCode
Subsample::backwardWithBias(vl::Tensor &derInput,
                            vl::Tensor &derBiases,
                            vl::Tensor const &derOutput) const
{
  return dispatch<SubsampleAndBiasBackward>()(*this,derInput,derBiases,derOutput) ;
}
